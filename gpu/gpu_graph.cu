#include "hip/hip_runtime.h"
/**
 * 这个版本里面没有细粒度计时。有计时的在gpu_graph_with_timer.cu里面。
 * 而且计时的方式与zms版本略有区别。
 */
#include <graph.h>
#include <dataloader.h>
#include <vertex_set.h>
#include <common.h>

#include <cassert>
#include <cstring>
#include <cstdint>
#include <string>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <>

#include <sys/time.h>
#include <chrono>

constexpr int THREADS_PER_BLOCK = 256;
constexpr int THREADS_PER_WARP = 32;
constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / THREADS_PER_WARP;

// 是否要用<chrono>中的内容进行替代？
class TimeInterval{
public:
    TimeInterval(){
        check();
    }

    void check(){
        gettimeofday(&tp, NULL);
    }

    void print(const char* title){
        struct timeval tp_end, tp_res;
        gettimeofday(&tp_end, NULL);
        timersub(&tp_end, &tp, &tp_res);
        printf("%s: %ld s %06ld us.\n", title, tp_res.tv_sec, tp_res.tv_usec);
    }
private:
    struct timeval tp;
};

TimeInterval allTime;
TimeInterval tmpTime;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#define get_edge_index(v, l, r) do { \
    l = vertex[v]; \
    r = vertex[v + 1]; \
} while(0)

template <typename T>
__device__ inline void swap(T& a, T& b)
{
    T t(std::move(a));
    a = std::move(b);
    b = std::move(t);
}

struct GPUGroupDim2 {
    int* data;
    int size;
};

struct GPUGroupDim1 {
    GPUGroupDim2* data;
    int size;
};

struct GPUGroupDim0 {
    GPUGroupDim1* data;
    int size;
};

class GPUSchedule {
public:
    __host__ void transform_in_exclusion_optimize_group_val(const Schedule& schedule)
    {
        /** @todo 注意当容斥优化无法使用时，内存分配会失败。需要修正 */
        int in_exclusion_optimize_num = schedule.get_in_exclusion_optimize_num();
        gpuErrchk( hipMallocManaged((void**)&in_exclusion_optimize_val, sizeof(int) * schedule.in_exclusion_optimize_val.size()));
        for (auto val : schedule.in_exclusion_optimize_val)
            in_exclusion_optimize_val[in_exclusion_optimize_val_size++] = val;
        in_exclusion_optimize_val_size = schedule.in_exclusion_optimize_val.size();
        
        //这部分有太多重复访存操作了（比如循环中的.data[i].data[j]，直接用一个tmp指针就行了），之后考虑优化掉（不过感觉O3会帮忙自动优化的）
        in_exclusion_optimize_group.size = schedule.in_exclusion_optimize_group.size();
        gpuErrchk( hipMallocManaged((void**)&in_exclusion_optimize_group.data, sizeof(GPUGroupDim1) * in_exclusion_optimize_group.size));
        for (int i = 0; i < schedule.in_exclusion_optimize_group.size(); ++i)
        {
            in_exclusion_optimize_group.data[i].size = schedule.in_exclusion_optimize_group[i].size();
            gpuErrchk( hipMallocManaged((void**)&in_exclusion_optimize_group.data[i].data, sizeof(GPUGroupDim2) * in_exclusion_optimize_group.data[i].size));
            for (int j = 0; j < schedule.in_exclusion_optimize_group[i].size(); ++j)
            {
                in_exclusion_optimize_group.data[i].data[j].size = schedule.in_exclusion_optimize_group[i][j].size();
                gpuErrchk( hipMallocManaged((void**)&in_exclusion_optimize_group.data[i].data[j].data, sizeof(int) * in_exclusion_optimize_group.data[i].data[j].size));
                for (int k = 0; k < schedule.in_exclusion_optimize_group[i][j].size(); ++k)
                    in_exclusion_optimize_group.data[i].data[j].data[k] = schedule.in_exclusion_optimize_group[i][j][k];
            }
        }
    }

    inline __device__ int get_total_prefix_num() const { return total_prefix_num;}
    inline __device__ int get_father_prefix_id(int prefix_id) const { return father_prefix_id[prefix_id];}
    inline __device__ int get_loop_set_prefix_id(int loop) const { return loop_set_prefix_id[loop];}
    inline __device__ int get_size() const { return size;}
    inline __device__ int get_last(int i) const { return last[i];}
    inline __device__ int get_next(int i) const { return next[i];}
    inline __device__ int get_in_exclusion_optimize_num() const { return in_exclusion_optimize_num;}
    inline __device__ int get_total_restrict_num() const { return total_restrict_num;}
    inline __device__ int get_restrict_last(int i) const { return restrict_last[i];}
    inline __device__ int get_restrict_next(int i) const { return restrict_next[i];}
    inline __device__ int get_restrict_index(int i) const { return restrict_index[i];}
    inline __device__ int get_k_val() const { return k_val;} // see below (the k_val's definition line) before using this function

    int* adj_mat;
    int* father_prefix_id;
    int* last;
    int* next;
    int* loop_set_prefix_id;
    int* restrict_last;
    int* restrict_next;
    int* restrict_index;
    int* in_exclusion_optimize_val;
    GPUGroupDim0 in_exclusion_optimize_group;
    int in_exclusion_optimize_val_size;
    int size;
    int total_prefix_num;
    int total_restrict_num;
    int in_exclusion_optimize_num;
    int k_val;
};

// __device__ void intersection1(uint32_t *tmp, uint32_t *lbases, uint32_t *rbases, uint32_t ln, uint32_t rn, uint32_t* p_tmp_size);
__device__ void intersection2(uint32_t *tmp, const uint32_t *lbases, const uint32_t *rbases, uint32_t ln, uint32_t rn, uint32_t* p_tmp_size);
static __device__ uint32_t do_intersection(uint32_t*, const uint32_t*, const uint32_t*, uint32_t, uint32_t);

class GPUVertexSet
{
public:
    __device__ GPUVertexSet()
    {
        size = 0;
        data = NULL;
    }
    __device__ int get_size() const { return size;}
    __device__ uint32_t get_data(int i) const { return data[i];}
    __device__ void push_back(uint32_t val) { data[size++] = val;}
    __device__ void pop_back() { --size;}
    __device__ uint32_t get_last() const {return data[size - 1];}
    __device__ void set_data_ptr(uint32_t* ptr) { data = ptr;}
    __device__ uint32_t* get_data_ptr() const { return data;}
    __device__ bool has_data (uint32_t val) const // 注意：这里不用二分，调用它的是较小的无序集合
    {
        for (int i = 0; i < size; ++i)
            if (data[i] == val)
                return true;
        return false;
    }
    __device__ void init() { size = 0; }
    __device__ void init(uint32_t input_size, uint32_t* input_data)
    {
        size = input_size;
        data = input_data; //之后如果把所有prefix放到shared memory，由于input data在global memory上（因为是原图的边集），所以改成memcpy
    }
    __device__ void copy_from(const GPUVertexSet& other)//考虑改为并行
    {
        // 这个版本可能会有bank conflict
        uint32_t input_size = other.get_size(), *input_data = other.get_data_ptr();
        size = input_size;
        int lid = threadIdx.x % THREADS_PER_WARP; // lane id
        int size_per_thread = (input_size + THREADS_PER_WARP - 1) / THREADS_PER_WARP;
        int start = size_per_thread * lid;
        int end = min(start + size_per_thread, input_size);
        for (int i = start; i < end; ++i)
            data[i] = input_data[i];
        __threadfence_block();
    }
    __device__ void build_vertex_set(const GPUSchedule* schedule, const GPUVertexSet* vertex_set, uint32_t* input_data, uint32_t input_size, int prefix_id)
    {
        int father_id = schedule->get_father_prefix_id(prefix_id);
        if (father_id == -1)
        {
            if (threadIdx.x % THREADS_PER_WARP == 0)
                init(input_size, input_data);
            __threadfence_block();
        }
        else
        {
            intersection2(this->data, vertex_set[father_id].get_data_ptr(), input_data, vertex_set[father_id].get_size(), input_size, &this->size);
        }
    }

    __device__ void intersection_with(const GPUVertexSet& other)
    {
        uint32_t ret = do_intersection(data, data, other.get_data_ptr(), size, other.get_size());
        if (threadIdx.x % THREADS_PER_WARP == 0)
            size = ret;
        __threadfence_block();
    }

private:
    uint32_t size;
    uint32_t* data;
};

__device__ unsigned long long dev_sum = 0;
__device__ unsigned int dev_cur_edge = 0;

/**
 * search-based intersection
 * 
 * returns the size of the intersection set
 * 
 * @note：a和b并不是地位相等的。如果要进行in-place操作，请把输入放在a而不是b。
 * @todo：shared memory缓存优化
 */
__device__ uint32_t do_intersection(uint32_t* out, const uint32_t* a, const uint32_t* b, uint32_t na, uint32_t nb)
{
    __shared__ uint32_t block_out_offset[THREADS_PER_BLOCK];
    __shared__ uint32_t block_out_size[WARPS_PER_BLOCK];

    int wid = threadIdx.x / THREADS_PER_WARP; // warp id
    int lid = threadIdx.x % THREADS_PER_WARP; // lane id
    uint32_t *out_offset = block_out_offset + wid * THREADS_PER_WARP;
    uint32_t &out_size = block_out_size[wid];

    if (lid == 0)
        out_size = 0;

    uint32_t num_done = 0;
    while (num_done < na) {
        bool found = 0;
        uint32_t u = 0;
        if (num_done + lid < na) {
            u = a[num_done + lid]; // u: an element in set a
            /*
            // 我这里这样写并没有变快，反而明显慢了
            int x, s[3], &l = s[0], &r = s[1], &mid = s[2];
            l = 0, r = int(nb) - 1, mid = (int(nb) - 1) >> 1;
            while (l <= r && !found) {
                uint32_t v = b[mid];
                found = (v == u);
                x = (v < u);
                mid += 2 * x - 1;
                swap(mid, s[!x]);
                mid = (l + r) >> 1;
            }
            */
            int mid, l = 0, r = int(nb) - 1;
            while (l <= r) {
                mid = (l + r) >> 1;
                if (b[mid] < u) {
                    l = mid + 1;
                } else if (b[mid] > u) {
                    r = mid - 1;
                } else {
                    found = 1;
                    break;
                }
            }
        }
        out_offset[lid] = found;
        __threadfence_block();

        #pragma unroll
        for (int s = 1; s < THREADS_PER_WARP; s *= 2) {
            uint32_t v = lid >= s ? out_offset[lid - s] : 0;
            // __threadfence_block(); // 在以warp为基本单位执行时，这句应该不需要吧？
            out_offset[lid] += v;
            __threadfence_block();
        }
        
        /*
        // work-efficient parallel scan，但常数大，实测速度不行
        #pragma unroll
        for (int s = 1; s < THREADS_PER_WARP; s <<= 1) {
            int i = (lid + 1) * s * 2 - 1;
            if (i < THREADS_PER_WARP)
                out_offset[i] += out_offset[i - s];
            __threadfence_block();
        }

        #pragma unroll
        for (int s = THREADS_PER_WARP / 4; s > 0; s >>= 1) {
            int i = (lid + 1) * s * 2 - 1;
            if ((i + s) < THREADS_PER_WARP)
                out_offset[i + s] += out_offset[i];
            __threadfence_block();
        }
        */
        
        if (found) {
            uint32_t offset = out_offset[lid] - 1;
            out[out_size + offset] = u;
        }

        if (lid == 0)
            out_size += out_offset[THREADS_PER_WARP - 1];

        /*
        // 使用warp shuffle的scan，但实测速度更不行
        uint32_t offset = found;
        #pragma unroll
        for (int i = 1; i < THREADS_PER_WARP; i *= 2) {
            uint32_t t = __shfl_up_sync(0xffffffff, offset, i);
            if (lid >= i)
                offset += t;
        }

        if (found)
            out[out_size + offset - 1] = u;
        if (lid == THREADS_PER_WARP - 1) // 总和被warp中最后一个线程持有
            out_size += offset;
        */

        num_done += THREADS_PER_WARP;
    }

    __threadfence_block();
    return out_size;
}

/**
 * wrapper of search based intersection `do_intersection`
 * 
 * 注意：不能进行in-place操作。若想原地操作则应当把交换去掉。
 */
__device__ void intersection2(uint32_t *tmp, const uint32_t *lbases, const uint32_t *rbases, uint32_t ln, uint32_t rn, uint32_t* p_tmp_size)
{
    // make sure ln <= rn
    if (ln > rn) {
        swap(ln, rn);
        swap(lbases, rbases);
    }
    /**
     * @todo 考虑ln < rn <= 32时，每个线程在lbases里面找rbases的一个元素可能会更快
     */

    uint32_t intersection_size = do_intersection(tmp, lbases, rbases, ln, rn);

    if (threadIdx.x % THREADS_PER_WARP == 0)
        *p_tmp_size = intersection_size;
    __threadfence_block();
}

/**
 * @brief calculate | set0 - set1 |
 * @note set0 should be an ordered set, while set1 can be unordered
 * @todo rename 'subtraction' => 'difference'
 */
__device__ int unordered_subtraction_size(const GPUVertexSet& set0, const GPUVertexSet& set1, int size_after_restrict = -1)
{
    __shared__ int block_ret[WARPS_PER_BLOCK];

    int size0 = set0.get_size();
    int size1 = set1.get_size();
    if (size_after_restrict != -1)
        size0 = size_after_restrict;

    int wid = threadIdx.x / THREADS_PER_WARP;
    int lid = threadIdx.x % THREADS_PER_WARP;
    int &ret = block_ret[wid];
    if (lid == 0)
        ret = size0;
    __threadfence_block();

    int done1 = 0;
    while (done1 < size1)
    {
        if (lid + done1 < size1)
        {
            int l = 0, r = size0 - 1;
            uint32_t val = set1.get_data(lid + done1);
            //考虑之后换一下二分查找的写法，比如改为l < r，然后把mid的判断从循环里去掉，放到循环外(即最后l==r的时候)
            while (l <= r)
            {
                int mid = (l + r) >> 1;
                if (set0.get_data(mid) == val)
                {
                    atomicSub(&ret, 1);
                    break;
                }
                if (set0.get_data(mid) < val)
                    l = mid + 1;
                else
                    r = mid - 1;
            }
            //binary search
        }
        done1 += THREADS_PER_WARP;
    }

    __threadfence_block();
    return ret;
}

/**
 * @brief 递归版本的pattern matching主函数。
 * @note 调用处初始深度为2（已经匹配了一条边对应的两个点）
 */
__device__ void GPU_pattern_matching_aggressive_func(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
    GPUVertexSet& tmp_set, unsigned long long& local_ans, int depth, uint32_t *edge, uint32_t *vertex)
{
    int loop_set_prefix_id = schedule->get_loop_set_prefix_id(depth);
    int loop_size = vertex_set[loop_set_prefix_id].get_size();
    if (loop_size <= 0)
        return;

    uint32_t* loop_data_ptr = vertex_set[loop_set_prefix_id].get_data_ptr();

    if( depth == schedule->get_size() - schedule->get_in_exclusion_optimize_num())
    {
        int in_exclusion_optimize_num = schedule->get_in_exclusion_optimize_num();
        //int* loop_set_prefix_ids[ in_exclusion_optimize_num ];
        int loop_set_prefix_ids[8]; // @todo 偷懒用了static，之后考虑改成dynamic
        // @todo 这里有硬编码的数字，之后考虑修改
        loop_set_prefix_ids[0] = loop_set_prefix_id;
        for(int i = 1; i < in_exclusion_optimize_num; ++i)
            loop_set_prefix_ids[i] = schedule->get_loop_set_prefix_id( depth + i );

        for(int optimize_rank = 0; optimize_rank < schedule->in_exclusion_optimize_group.size; ++optimize_rank) {
            const GPUGroupDim1& cur_graph = schedule->in_exclusion_optimize_group.data[optimize_rank];
            long long val = schedule->in_exclusion_optimize_val[optimize_rank];

            for(int cur_graph_rank = 0; cur_graph_rank < cur_graph.size; ++cur_graph_rank) {
                if(cur_graph.data[cur_graph_rank].size == 1) {
                    int id = loop_set_prefix_ids[cur_graph.data[cur_graph_rank].data[0]];
                    //val = val * unordered_subtraction_size(vertex_set[id], subtraction_set);
                    int tmp = unordered_subtraction_size(vertex_set[id], subtraction_set);
                    val = val * tmp;
                }
                else {
                    int id = loop_set_prefix_ids[cur_graph.data[cur_graph_rank].data[0]];
                    tmp_set.copy_from(vertex_set[id]);

                    for(int i = 1; i < cur_graph.data[cur_graph_rank].size; ++i) {
                        int id = loop_set_prefix_ids[cur_graph.data[cur_graph_rank].data[i]];
                        tmp_set.intersection_with(vertex_set[id]);
                    }
                    
                    int tmp = unordered_subtraction_size(tmp_set, subtraction_set);
                    val = val * tmp;
                }
                if (val == 0)
                    break;
            }

            local_ans += val;
        }
        return;
    }

    // 无容斥优化的最后一层
    if (depth == schedule->get_size() - 1)
    {
        //TODO
        assert(false);

        //if (threadIdx.x == 0)
        //    local_ans += val;
    }

    uint32_t min_vertex = 0xffffffff;
    for (int i = schedule->get_restrict_last(depth); i != -1; i = schedule->get_restrict_next(i))
        if (min_vertex > subtraction_set.get_data(schedule->get_restrict_index(i)))
            min_vertex = subtraction_set.get_data(schedule->get_restrict_index(i));
    for (int i = 0; i < loop_size; ++i)
    {
        uint32_t v = loop_data_ptr[i];
        if (min_vertex <= v)
            break;
        if (subtraction_set.has_data(v))
            continue;
        unsigned int l, r;
        get_edge_index(v, l, r);
        bool is_zero = false;
        for (int prefix_id = schedule->get_last(depth); prefix_id != -1; prefix_id = schedule->get_next(prefix_id))
        {
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
            if (vertex_set[prefix_id].get_size() == 0) {
                is_zero = true;
                break;
            }
        }
        if (is_zero)
            continue;
        if(threadIdx.x % THREADS_PER_WARP == 0)
            subtraction_set.push_back(v);
        __threadfence_block();
        GPU_pattern_matching_aggressive_func(schedule, vertex_set, subtraction_set, tmp_set, local_ans, depth + 1, edge, vertex);
        if(threadIdx.x % THREADS_PER_WARP == 0)
            subtraction_set.pop_back();
        __threadfence_block();
    }
}


/**
 * @brief 最终层的容斥原理优化计算。
 */
__device__ void GPU_pattern_matching_final_in_exclusion(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
    GPUVertexSet& tmp_set, unsigned long long& local_ans, int depth, uint32_t *edge, uint32_t *vertex)
{
    int in_exclusion_optimize_num = schedule->get_in_exclusion_optimize_num();
    //int* loop_set_prefix_ids[ in_exclusion_optimize_num ];
    int loop_set_prefix_ids[8];//偷懒用了static，之后考虑改成dynamic
    // 这里有硬编码的数字，之后考虑修改。
    loop_set_prefix_ids[0] = schedule->get_loop_set_prefix_id(depth);
    for(int i = 1; i < in_exclusion_optimize_num; ++i)
        loop_set_prefix_ids[i] = schedule->get_loop_set_prefix_id( depth + i );

    for (int optimize_rank = 0; optimize_rank < schedule->in_exclusion_optimize_group.size; ++optimize_rank) {
        const GPUGroupDim1& cur_graph = schedule->in_exclusion_optimize_group.data[optimize_rank];
        long long val = schedule->in_exclusion_optimize_val[optimize_rank];

        for (int cur_graph_rank = 0; cur_graph_rank < cur_graph.size; ++cur_graph_rank) {
            if (cur_graph.data[cur_graph_rank].size == 1) {
                int id = loop_set_prefix_ids[cur_graph.data[cur_graph_rank].data[0]];
                //val = val * unordered_subtraction_size(vertex_set[id], subtraction_set);
                int tmp = unordered_subtraction_size(vertex_set[id], subtraction_set);
                val = val * tmp;
            } else {
                int id = loop_set_prefix_ids[cur_graph.data[cur_graph_rank].data[0]];
                tmp_set.copy_from(vertex_set[id]);

                for (int i = 1; i < cur_graph.data[cur_graph_rank].size; ++i) {
                    int id = loop_set_prefix_ids[cur_graph.data[cur_graph_rank].data[i]];
                    tmp_set.intersection_with(vertex_set[id]);
                }

                int tmp = unordered_subtraction_size(tmp_set, subtraction_set);
                val = val * tmp;
            }
            if (val == 0)
                break;
        }

        local_ans += val;
    }
}

constexpr int MAX_DEPTH = 5; // 非递归pattern matching支持的最大深度

template <int depth>
__device__ void GPU_pattern_matching_func(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
    GPUVertexSet& tmp_set, unsigned long long& local_ans, uint32_t *edge, uint32_t *vertex)
{
    int loop_set_prefix_id = schedule->get_loop_set_prefix_id(depth);
    int loop_size = vertex_set[loop_set_prefix_id].get_size();
    if (loop_size <= 0)
        return;

    if (depth == schedule->get_size() - schedule->get_in_exclusion_optimize_num()) {
        GPU_pattern_matching_final_in_exclusion(schedule, vertex_set, subtraction_set, tmp_set, local_ans, depth, edge, vertex);
        return;    
    }

    uint32_t* loop_data_ptr = vertex_set[loop_set_prefix_id].get_data_ptr();
    uint32_t min_vertex = 0xffffffff;
    for (int i = schedule->get_restrict_last(depth); i != -1; i = schedule->get_restrict_next(i))
        if (min_vertex > subtraction_set.get_data(schedule->get_restrict_index(i)))
            min_vertex = subtraction_set.get_data(schedule->get_restrict_index(i));
    for (int i = 0; i < loop_size; ++i)
    {
        uint32_t v = loop_data_ptr[i];
        if (min_vertex <= v)
            break;
        if (subtraction_set.has_data(v))
            continue;
        unsigned int l, r;
        get_edge_index(v, l, r);
        bool is_zero = false;
        for (int prefix_id = schedule->get_last(depth); prefix_id != -1; prefix_id = schedule->get_next(prefix_id))
        {
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
            if (vertex_set[prefix_id].get_size() == 0) {
                is_zero = true;
                break;
            }
        }
        if (is_zero)
            continue;
        if (depth + 1 != MAX_DEPTH) {
            if (threadIdx.x % THREADS_PER_WARP == 0)
                subtraction_set.push_back(v);
            __threadfence_block();
        }
        GPU_pattern_matching_func<depth + 1>(schedule, vertex_set, subtraction_set, tmp_set, local_ans, edge, vertex);
        if (depth + 1 != MAX_DEPTH) {
            if (threadIdx.x % THREADS_PER_WARP == 0)
                subtraction_set.pop_back();
            __threadfence_block();
        }
    }
}

template <>
__device__ void GPU_pattern_matching_func<MAX_DEPTH>(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
    GPUVertexSet& tmp_set, unsigned long long& local_ans, uint32_t *edge, uint32_t *vertex)
{
    // assert(false);
}

/**
 * @note `buffer_size`实际上是每个节点的最大邻居数量，而非所用空间大小
 */
__global__ void gpu_pattern_matching(uint32_t edge_num, uint32_t buffer_size, uint32_t *edge_from, uint32_t *edge, uint32_t *vertex, uint32_t *tmp, const GPUSchedule* schedule) {
    __shared__ unsigned int block_edge_idx[WARPS_PER_BLOCK];
    //之后考虑把tmp buffer都放到shared里来（如果放得下）
    extern __shared__ GPUVertexSet block_vertex_set[];
    
    int num_prefixes = schedule->get_total_prefix_num();
    int num_vertex_sets_per_warp = num_prefixes + 2;

    int wid = threadIdx.x / THREADS_PER_WARP; // warp id within the block
    int lid = threadIdx.x % THREADS_PER_WARP; // lane id
    int global_wid = blockIdx.x * WARPS_PER_BLOCK + wid; // global warp id
    unsigned int &edge_idx = block_edge_idx[wid];
    GPUVertexSet *vertex_set = block_vertex_set + wid * num_vertex_sets_per_warp;

    if (lid == 0) {
        edge_idx = 0;
        uint32_t offset = buffer_size * global_wid * num_vertex_sets_per_warp;
        for (int i = 0; i < num_vertex_sets_per_warp; ++i)
        {
            vertex_set[i].set_data_ptr(tmp + offset); // 注意这是个指针+整数运算，自带*4
            offset += buffer_size;
        }
    }
    GPUVertexSet& subtraction_set = vertex_set[num_prefixes];
    GPUVertexSet& tmp_set = vertex_set[num_prefixes + 1];

    __threadfence_block(); //之后考虑把所有的syncthreads都改成syncwarp


    uint32_t v0, v1;
    uint32_t l, r;

    unsigned long long sum = 0;

    while (true) {
        if (lid == 0) {
            //if(++edgeI >= edgeEnd) { //这个if语句应该是每次都会发生吧？（是的
                edge_idx = atomicAdd(&dev_cur_edge, 1);
                //edgeEnd = min(edge_num, edgeI + 1); //这里不需要原子读吗
                unsigned int i = edge_idx;
                if (i < edge_num)
                {
                    subtraction_set.init();
                    subtraction_set.push_back(edge_from[i]);
                    subtraction_set.push_back(edge[i]);
                }
            //}
        }

        __threadfence_block();

        unsigned int i = edge_idx;
        if(i >= edge_num) break;
       
       // for edge in E
        v0 = edge_from[i];
        v1 = edge[i];

        bool is_zero = false;
        get_edge_index(v0, l, r);
        for (int prefix_id = schedule->get_last(0); prefix_id != -1; prefix_id = schedule->get_next(prefix_id))
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);

        //目前只考虑pattern size>2的情况
        //start v1, depth = 1
        if (schedule->get_restrict_last(1) != -1 && v0 <= v1)
            continue;
        
        get_edge_index(v1, l, r);
        for (int prefix_id = schedule->get_last(1); prefix_id != -1; prefix_id = schedule->get_next(prefix_id))
        {
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
            if (vertex_set[prefix_id].get_size() == 0) {
                is_zero = true;
                break;
            }
        }
        if (is_zero)
            continue;
        
        unsigned long long local_sum = 0; // local sum (corresponding to an edge index)
        GPU_pattern_matching_func<2>(schedule, vertex_set, subtraction_set, tmp_set, local_sum, edge, vertex);
        // GPU_pattern_matching_aggressive_func(schedule, vertex_set, subtraction_set, tmp_set, local_sum, 2, edge, vertex);
        sum += local_sum;
    }

    if (lid == 0) {
        atomicAdd(&dev_sum, sum);
    }
}

void pattern_matching_init(Graph *g, const Schedule& schedule) {
    int num_blocks = 4096;
    int num_total_warps = num_blocks * WARPS_PER_BLOCK;

    size_t size_edge = g->e_cnt * sizeof(uint32_t);
    size_t size_vertex = (g->v_cnt + 1) * sizeof(uint32_t);
    size_t size_tmp = VertexSet::max_intersection_size * sizeof(uint32_t) * num_total_warps * (schedule.get_total_prefix_num() + 2); //prefix + subtraction + tmp

    schedule.print_schedule();
    uint32_t *edge_from = new uint32_t[g->e_cnt];
    for(uint32_t i = 0; i < g->v_cnt; ++i)
        for(uint32_t j = g->vertex[i]; j < g->vertex[i+1]; ++j)
            edge_from[j] = i;

    tmpTime.check(); 

    uint32_t *dev_edge;
    uint32_t *dev_edge_from;
    uint32_t *dev_vertex;
    uint32_t *dev_tmp;

    gpuErrchk( hipMalloc((void**)&dev_edge, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_edge_from, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_vertex, size_vertex));
    gpuErrchk( hipMalloc((void**)&dev_tmp, size_tmp));

    gpuErrchk( hipMemcpy(dev_edge, g->edge, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_edge_from, edge_from, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_vertex, g->vertex, size_vertex, hipMemcpyHostToDevice));

    unsigned long long sum = 0;

    //memcpy schedule
    GPUSchedule* dev_schedule;
    gpuErrchk( hipMallocManaged((void**)&dev_schedule, sizeof(GPUSchedule)));
    dev_schedule->transform_in_exclusion_optimize_group_val(schedule);
    int schedule_size = schedule.get_size();
    int max_prefix_num = schedule_size * (schedule_size - 1) / 2;
    gpuErrchk( hipMallocManaged((void**)&dev_schedule->adj_mat, sizeof(int) * schedule_size * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->adj_mat, schedule.get_adj_mat_ptr(), sizeof(int) * schedule_size * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->father_prefix_id, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->father_prefix_id, schedule.get_father_prefix_id_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->last, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->last, schedule.get_last_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->next, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->next, schedule.get_next_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->loop_set_prefix_id, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->loop_set_prefix_id, schedule.get_loop_set_prefix_id_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_last, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_last, schedule.get_restrict_last_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));
    
    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_next, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_next, schedule.get_restrict_next_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));
    
    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_index, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_index, schedule.get_restrict_index_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    dev_schedule->size = schedule.get_size();
    dev_schedule->total_prefix_num = schedule.get_total_prefix_num();
    dev_schedule->total_restrict_num = schedule.get_total_restrict_num();
    dev_schedule->in_exclusion_optimize_num = schedule.get_in_exclusion_optimize_num();
    dev_schedule->k_val = schedule.get_k_val();

    printf("schedule.prefix_num: %d\n", schedule.get_total_prefix_num());
    printf("shared memory for vertex set per block: %ld bytes\n", 
        (schedule.get_total_prefix_num() + 2) * WARPS_PER_BLOCK * sizeof(GPUVertexSet));

    tmpTime.print("Prepare time cost");
    tmpTime.check();

    uint32_t buffer_size = VertexSet::max_intersection_size;
    uint32_t block_shmem_size = (schedule.get_total_prefix_num() + 2) * WARPS_PER_BLOCK * sizeof(GPUVertexSet);
    // 注意：此处没有错误，buffer_size代指每个顶点集所需的int数目，无需再乘sizeof(uint32_t)，但是否考虑对齐？
    //因为目前用了managed开内存，所以第一次运行kernel会有一定额外开销，考虑运行两次，第一次作为warmup
    gpu_pattern_matching<<<num_blocks, THREADS_PER_BLOCK, block_shmem_size>>>
        (g->e_cnt, buffer_size, dev_edge_from, dev_edge, dev_vertex, dev_tmp, dev_schedule);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    gpuErrchk( hipMemcpyFromSymbol(&sum, HIP_SYMBOL(dev_sum), sizeof(sum)) );

    printf("count %llu\n", sum);
    tmpTime.print("Counting time cost");
    //之后需要加上hipFree

    // 尝试释放一些内存
    gpuErrchk(hipFree(dev_edge));
    gpuErrchk(hipFree(dev_edge_from));
    gpuErrchk(hipFree(dev_vertex));
    gpuErrchk(hipFree(dev_tmp));

    gpuErrchk(hipFree(dev_schedule->adj_mat));
    gpuErrchk(hipFree(dev_schedule->father_prefix_id));
    gpuErrchk(hipFree(dev_schedule->last));
    gpuErrchk(hipFree(dev_schedule->next));
    gpuErrchk(hipFree(dev_schedule->loop_set_prefix_id));
    gpuErrchk(hipFree(dev_schedule->restrict_last));
    gpuErrchk(hipFree(dev_schedule->restrict_next));
    gpuErrchk(hipFree(dev_schedule->restrict_index));
    gpuErrchk(hipFree(dev_schedule));

    delete[] edge_from;
}

int main(int argc,char *argv[]) {
    Graph *g;
    DataLoader D;

    if (argc < 2) {
        printf("Usage: %s dataset_name graph_file [binary/text]\n", argv[0]);
        printf("Example: %s Patents ~hzx/data/patents_bin binary\n", argv[0]);
        printf("Example: %s Patents ~zms/patents_input\n", argv[0]);

        printf("\nExperimental usage: %s [graph_file.g]\n", argv[0]);
        printf("Example: %s ~hzx/data/patents.g\n", argv[0]);
        return 0;
    }

    bool binary_input = false;
    if (argc >= 4)
        binary_input = (strcmp(argv[3], "binary") == 0);

    DataType my_type;
    if (argc >= 3) {
        GetDataType(my_type, argv[1]);

        if (my_type == DataType::Invalid) {
            printf("Dataset not found!\n");
            return 0;
        }
    }

    using std::chrono::system_clock;
    auto t1 = system_clock::now();

    bool ok;
    if (argc >= 3) {
        // 注：load_data的第四个参数用于指定是否读取二进制文件输入，默认为false
        ok = D.load_data(g, my_type, argv[2], binary_input);
    } else {
        ok = D.fast_load(g, argv[1]);
    }
    if (!ok) {
        printf("data load failure :-(\n");
        return 0;
    }

    auto t2 = system_clock::now();
    auto load_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    printf("Load data success! time: %g seconds\n", load_time.count() / 1.0e6);
    fflush(stdout);

    allTime.check();

    const char *pattern_str = "0111010011100011100001100"; // 5 house p1
    // const char *pattern_str = "011011101110110101011000110000101000"; // 6 p2
    // const char *pattern_str = "0111111101111111011101110100111100011100001100000"; // 7 p5
    // const char *pattern_str = "0111111101111111011001110100111100011000001100000"; // 7 p6

    Pattern p(5, pattern_str);
    printf("pattern = \n");
    p.print();
    printf("max intersection size %d\n", VertexSet::max_intersection_size);
    bool is_pattern_valid;
    bool use_in_exclusion_optimize = true;
    Schedule schedule(p, is_pattern_valid, 1, 1, use_in_exclusion_optimize, g->v_cnt, g->e_cnt, g->tri_cnt);
    //Schedule schedule(p, is_pattern_valid, 0, 1, use_in_exclusion_optimize, g->v_cnt, g->e_cnt, g->tri_cnt); // use the best schedule

    if (!is_pattern_valid) {
        printf("pattern is invalid!\n");
        return 0;
    }

    pattern_matching_init(g, schedule);

    allTime.print("Total time cost");

    return 0;
}
