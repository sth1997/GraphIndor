#include "hip/hip_runtime.h"
#include <schedule_IEP.h>
#include <dataloader.h>
#include <graph.h>
#include <mpi.h>
#include <omp.h>

#include <unistd.h>
#include <cstdlib>
#include <cassert>
#include <cstdio>
#include <atomic>
#include <tuple>
#include <utility>
#include <string>
#include <chrono>
#include <cmath>

#include "pattern_matching.cuh"

__device__ void gpu_pattern_matching_final_in_exclusion(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
    GPUVertexSet& tmp_set, unsigned long long& local_ans,  uint32_t *edge, uint32_t *vertex) {
    int last_pos = -1;
    long long val;

    extern __shared__ char ans_array[];
    int* ans = ((int*) (ans_array + schedule->ans_array_offset)) 
        + schedule->in_exclusion_optimize_vertex_id_size * (threadIdx.x / THREADS_PER_WARP);
    
    for (int i = 0; i < schedule->in_exclusion_optimize_vertex_id_size; ++i) {
        if (schedule->in_exclusion_optimize_vertex_flag[i]) {
            ans[i] = vertex_set[schedule->in_exclusion_optimize_vertex_id[i]].get_size() - schedule->in_exclusion_optimize_vertex_coef[i];
        } else {
            ans[i] = unordered_subtraction_size(vertex_set[schedule->in_exclusion_optimize_vertex_id[i]], subtraction_set);
        }
    }

    for (int pos = 0; pos < schedule->in_exclusion_optimize_array_size; ++pos) {
        if (pos == last_pos + 1)
            val = ans[schedule->in_exclusion_optimize_ans_pos[pos]];
        else {
            if (val != 0)
                val = val * ans[schedule->in_exclusion_optimize_ans_pos[pos]];
        }
        if (schedule->in_exclusion_optimize_flag[pos]) {
            last_pos = pos;
            local_ans += val * schedule->in_exclusion_optimize_coef[pos];
        }
    }
}

constexpr int MAX_DEPTH = 5; // 非递归pattern matching支持的最大深度

template <int depth>
__device__ void gpu_pattern_matching_func(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
    GPUVertexSet& tmp_set, unsigned long long& local_ans, uint32_t *edge, uint32_t *vertex)
{

    if (depth == schedule->get_size() - schedule->get_in_exclusion_optimize_num()) {
        gpu_pattern_matching_final_in_exclusion(schedule, vertex_set, subtraction_set, tmp_set, local_ans,  edge, vertex);
        return;    
    }

    int loop_set_prefix_id = schedule->get_loop_set_prefix_id(depth);
    int loop_size = vertex_set[loop_set_prefix_id].get_size();
    if (loop_size <= 0) //这个判断可能可以删了
        return;

    uint32_t* loop_data_ptr = vertex_set[loop_set_prefix_id].get_data_ptr();
    uint32_t min_vertex = 0xffffffff;
    for (int i = schedule->get_restrict_last(depth); i != -1; i = schedule->get_restrict_next(i))
        if (min_vertex > subtraction_set.get_data(schedule->get_restrict_index(i)))
            min_vertex = subtraction_set.get_data(schedule->get_restrict_index(i));
    for (int i = 0; i < loop_size; ++i) {
        uint32_t v = loop_data_ptr[i];
        if (min_vertex <= v)
            break;
        if (subtraction_set.has_data(v))
            continue;
        unsigned int l, r;
        get_edge_index(v, l, r);
        bool is_zero = false;
        for (int prefix_id = schedule->get_last(depth); prefix_id != -1; prefix_id = schedule->get_next(prefix_id)) {
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
            if (vertex_set[prefix_id].get_size() == schedule->get_break_size(prefix_id)) {
                is_zero = true;
                break;
            }
        }
        if (is_zero)
            continue;
        if (depth + 1 != MAX_DEPTH) {
            if (threadIdx.x % THREADS_PER_WARP == 0)
                subtraction_set.push_back(v);
            __threadfence_block();
        }
        gpu_pattern_matching_func<depth + 1>(schedule, vertex_set, subtraction_set, tmp_set, local_ans, edge, vertex);
        if (depth + 1 != MAX_DEPTH) {
            if (threadIdx.x % THREADS_PER_WARP == 0)
                subtraction_set.pop_back();
            __threadfence_block();
        }
    }
}

template <>
__device__ void gpu_pattern_matching_func<MAX_DEPTH>(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
        GPUVertexSet& tmp_set, unsigned long long& local_ans, uint32_t *edge, uint32_t *vertex)
{
    // assert(false);
}

// device global variables
__device__ unsigned long long dev_sum;
__device__ unsigned int dev_cur_task;

__global__ void gpu_pattern_matching(unsigned task_end, size_t buffer_size, uint32_t *edge_from, uint32_t *edge, uint32_t *vertex, uint32_t *tmp, const GPUSchedule* schedule) {
    __shared__ unsigned int block_edge_idx[WARPS_PER_BLOCK];
    extern __shared__ GPUVertexSet block_vertex_set[];
    
    int num_prefixes = schedule->get_total_prefix_num();
    int num_vertex_sets_per_warp = num_prefixes + 2;

    int wid = threadIdx.x / THREADS_PER_WARP; // warp id within the block
    int lid = threadIdx.x % THREADS_PER_WARP; // lane id
    int global_wid = blockIdx.x * WARPS_PER_BLOCK + wid; // global warp id
    unsigned int &edge_idx = block_edge_idx[wid];
    GPUVertexSet *vertex_set = block_vertex_set + wid * num_vertex_sets_per_warp;

    if (lid == 0) {
        edge_idx = 0;
        ptrdiff_t offset = buffer_size * global_wid * num_vertex_sets_per_warp;
        for (int i = 0; i < num_vertex_sets_per_warp; ++i) {
            vertex_set[i].set_data_ptr(tmp + offset); // 注意这是个指针+整数运算，自带*4
            offset += buffer_size;
        }
    }
    GPUVertexSet& subtraction_set = vertex_set[num_prefixes];
    GPUVertexSet& tmp_set = vertex_set[num_prefixes + 1];

    __threadfence_block();

    uint32_t v0, v1;
    uint32_t l, r;

    unsigned long long sum = 0;

    while (true) {
        if (lid == 0) {
            edge_idx = atomicAdd(&dev_cur_task, 1);
            unsigned int i = edge_idx;
            if (i < task_end) {
                subtraction_set.init();
                subtraction_set.push_back(edge_from[i]);
                subtraction_set.push_back(edge[i]);
            }
        }
        __threadfence_block();

        unsigned int i = edge_idx;
        if (i >= task_end) break;
       
        // for edge in E
        v0 = edge_from[i];
        v1 = edge[i];

        //目前只考虑pattern size>2的情况
        //start v1, depth = 1
        if (schedule->get_restrict_last(1) != -1 && v0 <= v1)
            continue;

        bool is_zero = false;
        get_edge_index(v0, l, r);
        for (int prefix_id = schedule->get_last(0); prefix_id != -1; prefix_id = schedule->get_next(prefix_id))
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
        
        get_edge_index(v1, l, r);
        for (int prefix_id = schedule->get_last(1); prefix_id != -1; prefix_id = schedule->get_next(prefix_id)) {
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
            if (vertex_set[prefix_id].get_size() == 0 && prefix_id < schedule->get_basic_prefix_num()) {
                is_zero = true;
                break;
            }
        }
        if (is_zero)
            continue;
        
        unsigned long long local_sum = 0; // local sum (corresponding to an edge index)
        gpu_pattern_matching_func<2>(schedule, vertex_set, subtraction_set, tmp_set, local_sum, edge, vertex);
        sum += local_sum;
    }

    if (lid == 0) {
        atomicAdd(&dev_sum, sum);
    }
}

struct GPUContext {
    int nr_blocks, nr_total_warps, block_shmem_size;
    uint32_t *dev_edge, *dev_edge_from, *dev_vertex, *dev_tmp;
    GPUSchedule *dev_schedule;    
};

void init_gpu_schedule(GPUContext& ctx, const Schedule_IEP& sched) {
    GPUSchedule *dev_schedule;
    gpu_check( hipMallocManaged((void**)&dev_schedule, sizeof(GPUSchedule)) );

    int n = sched.get_size();
    int max_prefix_num = n * (n - 1) / 2;
    
    auto only_need_size = new bool[max_prefix_num];
    for (int i = 0; i < max_prefix_num; ++i)
        only_need_size[i] = sched.get_prefix_only_need_size(i);

    int in_exclusion_optimize_vertex_id_size = sched.in_exclusion_optimize_vertex_id.size();
    int in_exclusion_optimize_array_size = sched.in_exclusion_optimize_coef.size();

    auto in_exclusion_optimize_vertex_id = &(sched.in_exclusion_optimize_vertex_id[0]);
    auto in_exclusion_optimize_vertex_coef = &(sched.in_exclusion_optimize_vertex_coef[0]);
    auto in_exclusion_optimize_vertex_flag = new bool[in_exclusion_optimize_vertex_id_size]; 

    auto in_exclusion_optimize_coef = &(sched.in_exclusion_optimize_coef[0]);
    auto in_exclusion_optimize_ans_pos = &(sched.in_exclusion_optimize_ans_pos[0]);
    auto in_exclusion_optimize_flag = new bool[in_exclusion_optimize_array_size];

    for (int i = 0; i < in_exclusion_optimize_vertex_id_size; ++i)
        in_exclusion_optimize_vertex_flag[i] = sched.in_exclusion_optimize_vertex_flag[i];
    
    for (int i = 0; i < in_exclusion_optimize_array_size; ++i)
        in_exclusion_optimize_flag[i] = sched.in_exclusion_optimize_flag[i];

    gpu_check( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_id, sizeof(int) * in_exclusion_optimize_vertex_id_size));
    gpu_check( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_id, in_exclusion_optimize_vertex_id, sizeof(int) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));
    
    gpu_check( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_flag, sizeof(bool) * in_exclusion_optimize_vertex_id_size));
    gpu_check( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_flag, in_exclusion_optimize_vertex_flag, sizeof(bool) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));
    
    gpu_check( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_coef, sizeof(int) * in_exclusion_optimize_vertex_id_size));
    gpu_check( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_coef, in_exclusion_optimize_vertex_coef, sizeof(int) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));

    gpu_check( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_coef, sizeof(int) * in_exclusion_optimize_array_size));
    gpu_check( hipMemcpy(dev_schedule->in_exclusion_optimize_coef, in_exclusion_optimize_coef, sizeof(int) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));

    gpu_check( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_flag, sizeof(bool) * in_exclusion_optimize_array_size));
    gpu_check( hipMemcpy(dev_schedule->in_exclusion_optimize_flag, in_exclusion_optimize_flag, sizeof(bool) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));
    
    gpu_check( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_ans_pos, sizeof(int) * in_exclusion_optimize_array_size));
    gpu_check( hipMemcpy(dev_schedule->in_exclusion_optimize_ans_pos, in_exclusion_optimize_ans_pos, sizeof(int) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));

    gpu_check( hipMallocManaged((void**)&dev_schedule->adj_mat, sizeof(int) * n * n));
    gpu_check( hipMemcpy(dev_schedule->adj_mat, sched.get_adj_mat_ptr(), sizeof(int) * n * n, hipMemcpyHostToDevice));

    gpu_check( hipMallocManaged((void**)&dev_schedule->father_prefix_id, sizeof(int) * max_prefix_num));
    gpu_check( hipMemcpy(dev_schedule->father_prefix_id, sched.get_father_prefix_id_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpu_check( hipMallocManaged((void**)&dev_schedule->last, sizeof(int) * n));
    gpu_check( hipMemcpy(dev_schedule->last, sched.get_last_ptr(), sizeof(int) * n, hipMemcpyHostToDevice));

    gpu_check( hipMallocManaged((void**)&dev_schedule->next, sizeof(int) * max_prefix_num));
    gpu_check( hipMemcpy(dev_schedule->next, sched.get_next_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));
    
    gpu_check( hipMallocManaged((void**)&dev_schedule->only_need_size, sizeof(bool) * max_prefix_num));
    gpu_check( hipMemcpy(dev_schedule->only_need_size, only_need_size, sizeof(bool) * max_prefix_num, hipMemcpyHostToDevice));
    
    gpu_check( hipMallocManaged((void**)&dev_schedule->break_size, sizeof(int) * max_prefix_num));
    gpu_check( hipMemcpy(dev_schedule->break_size, sched.get_break_size_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpu_check( hipMallocManaged((void**)&dev_schedule->loop_set_prefix_id, sizeof(int) * n));
    gpu_check( hipMemcpy(dev_schedule->loop_set_prefix_id, sched.get_loop_set_prefix_id_ptr(), sizeof(int) * n, hipMemcpyHostToDevice));

    gpu_check( hipMallocManaged((void**)&dev_schedule->restrict_last, sizeof(int) * n));
    gpu_check( hipMemcpy(dev_schedule->restrict_last, sched.get_restrict_last_ptr(), sizeof(int) * n, hipMemcpyHostToDevice));
    
    gpu_check( hipMallocManaged((void**)&dev_schedule->restrict_next, sizeof(int) * max_prefix_num));
    gpu_check( hipMemcpy(dev_schedule->restrict_next, sched.get_restrict_next_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));
    
    gpu_check( hipMallocManaged((void**)&dev_schedule->restrict_index, sizeof(int) * max_prefix_num));
    gpu_check( hipMemcpy(dev_schedule->restrict_index, sched.get_restrict_index_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    dev_schedule->in_exclusion_optimize_array_size = in_exclusion_optimize_array_size;
    dev_schedule->in_exclusion_optimize_vertex_id_size = in_exclusion_optimize_vertex_id_size;
    dev_schedule->size = n;
    dev_schedule->total_prefix_num = sched.get_total_prefix_num();
    dev_schedule->basic_prefix_num = sched.get_basic_prefix_num();
    dev_schedule->total_restrict_num = sched.get_total_restrict_num();
    dev_schedule->in_exclusion_optimize_num = sched.get_in_exclusion_optimize_num();

    uint32_t block_shmem_size = (sched.get_total_prefix_num() + 2) * WARPS_PER_BLOCK 
        * sizeof(GPUVertexSet) + in_exclusion_optimize_vertex_id_size * WARPS_PER_BLOCK * sizeof(int);
    dev_schedule->ans_array_offset = block_shmem_size - in_exclusion_optimize_vertex_id_size * WARPS_PER_BLOCK * sizeof(int);

    ctx.block_shmem_size = block_shmem_size;
    ctx.dev_schedule = dev_schedule;

    delete[] only_need_size;
    delete[] in_exclusion_optimize_vertex_flag;
    delete[] in_exclusion_optimize_flag;
}

void init_gpu_context(GPUContext& ctx, Graph* g, const Schedule_IEP& schedule) {
    ctx.nr_blocks = 1024;
    ctx.nr_total_warps = ctx.nr_blocks * WARPS_PER_BLOCK;

    size_t size_edge = g->e_cnt * sizeof(uint32_t);
    size_t size_vertex = (g->v_cnt + 1) * sizeof(uint32_t);
    size_t size_tmp = VertexSet::max_intersection_size * sizeof(uint32_t) 
        * ctx.nr_total_warps * (schedule.get_total_prefix_num() + 2); //prefix + subtraction + tmp

    gpu_check( hipMalloc((void**)&ctx.dev_edge, size_edge));
    gpu_check( hipMalloc((void**)&ctx.dev_edge_from, size_edge));
    gpu_check( hipMalloc((void**)&ctx.dev_vertex, size_vertex));
    gpu_check( hipMalloc((void**)&ctx.dev_tmp, size_tmp));

    gpu_check( hipMemcpy(ctx.dev_edge, g->edge, size_edge, hipMemcpyHostToDevice));
    gpu_check( hipMemcpy(ctx.dev_edge_from, g->edge_from, size_edge, hipMemcpyHostToDevice));
    gpu_check( hipMemcpy(ctx.dev_vertex, g->vertex, size_vertex, hipMemcpyHostToDevice));

    init_gpu_schedule(ctx, schedule);
}

void free_gpu_context(GPUContext& ctx) {
    // TODO
}

struct SpinLock {
    std::atomic_flag flag;

    SpinLock() : flag{ATOMIC_FLAG_INIT} {}
    void lock() { while (flag.test_and_set()) asm volatile ("pause"); }
    void unlock() { flag.clear(); }
};

template <typename Lock>
struct LockGuard {
    Lock &_lock;
    LockGuard(Lock &lock) : _lock{lock} { _lock.lock(); }
    ~LockGuard() { _lock.unlock(); }
};

__global__ void cuda_kernel(int node)
{
    printf("hello from cuda thread=%d block=%d got rank=%d\n", threadIdx.x, blockIdx.x, node);
}

__global__ void spin_kernel(clock_t cycles)
{
    clock_t start = clock64();
    while (clock64() - start < cycles)
        ;
    printf("spin_kernel done. thread=%d\n", threadIdx.x);
}

// #define log(fmt, ...) fprintf(stderr, fmt, ##__VA_ARGS__)
#define log(fmt, ...) (void)(fmt)

// shared by all threads
int nr_tasks;
using TaskStatus = std::tuple<int, int>;
TaskStatus task_status;
SpinLock task_status_lock;
volatile bool workers_should_stop = false;
volatile bool task_requested = false;
bool *workers_idle;

constexpr int CPU_WORKER_TASK_GRANULARITY = 10;
constexpr int GPU_WORKER_TASK_GRANULARITY = 5000;
constexpr int NODE_TASK_GRANULARUTY = 10000;
constexpr int INITIAL_NODE_TASKS = 10000;

enum MessageType {
    MSG_REQUEST_WORK,  // slave -> master
    MSG_DISPATCH_WORK, // master -> slave
    MSG_REPORT_ANS     // slave -> master
};

enum NodeState {
    WORKING, // normal working state
    WAITING  // no more work from master node, waiting for worker threads to finish
};

NodeState state = WORKING; // only used by scheduler thread
int global_cur_task; // only scheduler thread of master node will modify this var in working phase
int nr_idle_nodes = 0;
uint64_t global_ans = 0, gpu_ans = 0;
std::atomic<uint64_t> node_ans{0};

void process_message(uint64_t recv_buf[], uint64_t send_buf[], int node, int sender)
{
    MPI_Request send_req;
    switch (recv_buf[0]) {
    case MSG_REQUEST_WORK: { // me: master
        send_buf[0] = MSG_DISPATCH_WORK;
        send_buf[1] = global_cur_task;
        send_buf[2] = global_cur_task = std::min(global_cur_task + NODE_TASK_GRANULARUTY, nr_tasks);
        MPI_Isend(send_buf, 3, MPI_UINT64_T, sender, 0, MPI_COMM_WORLD, &send_req);
        log("master got work request from node %d, replying [%ld, %ld)\n", sender, send_buf[1], send_buf[2]);
        break;
    }
    case MSG_DISPATCH_WORK: { // me: slave
        int new_task_cur = recv_buf[1];
        int new_task_end = recv_buf[2];
        if (new_task_cur >= nr_tasks) {
            state = NodeState::WAITING;
            log("slave node %d enters WAITING state\n", node);
        } else {
            LockGuard<SpinLock> guard{task_status_lock};
            task_status = std::make_tuple(new_task_cur, new_task_end);
            log("slave node %d got task [%d, %d)\n", node, new_task_cur, new_task_end);
            task_requested = false;
        }
        break;
    }
    case MSG_REPORT_ANS: { // me: master
        ++nr_idle_nodes;
        global_ans += recv_buf[1];
        log("master receive answer %ld from node %d\n", recv_buf[1], sender);
        break;
    }
    }
}

// TODO: require lock?
bool all_workers_idle()
{
    int nr_threads = omp_get_max_threads();
    int idle_count = 0;
    for (int i = 0; i < nr_threads; ++i)
        if (workers_idle[i])
            ++idle_count;
    return idle_count == nr_threads;
}

// TODO: returns true when too many worker threads are idle?
bool should_request_work()
{
    LockGuard<SpinLock> guard{task_status_lock};
    return std::get<0>(task_status) >= std::get<1>(task_status);
}

// returns whether task status is successfully updated
bool update_task_range(std::tuple<int, int>& task_range, int max_nr_tasks)
{
    int task_cur, task_end, new_task_cur;
    LockGuard<SpinLock> guard{task_status_lock};
    std::tie(task_cur, task_end) = task_status;
    if (task_cur < task_end) {
        new_task_cur = std::min(task_cur + max_nr_tasks, task_end);
        task_range = std::make_tuple(task_cur, new_task_cur);
        task_status = std::make_tuple(new_task_cur, task_end);
        return true;
    }
    return false;
}

void launch_pattern_matching_kernel(const GPUContext& ctx, const TaskStatus& task_range) {
    int task_cur = std::get<0>(task_range);
    int task_end = std::get<1>(task_range);
    unsigned long long sum = 0;
    gpu_check( hipMemcpyToSymbol(HIP_SYMBOL(dev_sum), &sum, sizeof(sum)) );
    gpu_check( hipMemcpyToSymbol(HIP_SYMBOL(dev_cur_task), &task_cur, sizeof(task_cur)) );
    gpu_pattern_matching<<<ctx.nr_blocks, THREADS_PER_BLOCK, ctx.block_shmem_size>>>(
        task_end, VertexSet::max_intersection_size, ctx.dev_edge_from,
        ctx.dev_edge, ctx.dev_vertex, ctx.dev_tmp, ctx.dev_schedule
    );
}

// thread 0 is scheduler
void scheduler_loop(Graph* g, const Schedule_IEP& sched, int node, int comm_sz)
{
    GPUContext ctx;
    init_gpu_context(ctx, g, sched);

    hipEvent_t event;
    hipEventCreate(&event);
    TaskStatus gpu_task_range;
    bool gpu_working = false;

    constexpr int MSG_BUF_LEN = 256;
    static uint64_t recv_buf[MSG_BUF_LEN], send_buf[MSG_BUF_LEN];
    MPI_Request send_req, recv_req;
    MPI_Status mpi_status;

    MPI_Irecv(recv_buf, MSG_BUF_LEN, MPI_UINT64_T, MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, &recv_req);
    while (true) {
        if (node == 0 || state == NodeState::WORKING) {
            int msg_received = 0;
            MPI_Test(&recv_req, &msg_received, &mpi_status);
            if (msg_received) {
                process_message(recv_buf, send_buf, node, mpi_status.MPI_SOURCE);
                MPI_Irecv(recv_buf, MSG_BUF_LEN, MPI_UINT64_T, MPI_ANY_SOURCE, 0, MPI_COMM_WORLD, &recv_req);
            }
        }
        
        switch (state) {
        case NodeState::WORKING: {
            if (should_request_work()) {
                if (node != 0) {
                    if (!task_requested) {
                        send_buf[0] = MSG_REQUEST_WORK;
                        MPI_Isend(send_buf, 1, MPI_UINT64_T, 0, 0, MPI_COMM_WORLD, &send_req);
                        task_requested = true;
                    }
                } else {
                    int new_task_cur, new_task_end;
                    new_task_cur = global_cur_task;
                    new_task_end = global_cur_task = std::min(global_cur_task + NODE_TASK_GRANULARUTY, nr_tasks);
                    if (new_task_cur >= nr_tasks) {
                        state = NodeState::WAITING;
                        log("master node enters WAITING state\n");
                    } else {
                        LockGuard<SpinLock> guard{task_status_lock};
                        task_status = std::make_tuple(new_task_cur, new_task_end);
                        log("master node got task [%d, %d)\n", new_task_cur, new_task_end);
                    }
                }
            }
            break;
        }
        case NodeState::WAITING: {
            if (all_workers_idle()) {
                workers_should_stop = true;
                if (node != 0) {
                    send_buf[0] = MSG_REPORT_ANS;
                    send_buf[1] = node_ans;
                    MPI_Isend(send_buf, 2, MPI_UINT64_T, 0, 0, MPI_COMM_WORLD, &send_req);
                    return;
                } else {
                    if (nr_idle_nodes == comm_sz - 1)
                        return;
                }
            }
            break;
        }
        }

        if (!gpu_working) {
            if (update_task_range(gpu_task_range, GPU_WORKER_TASK_GRANULARITY)) {
                gpu_working = true;
                workers_idle[0] = false;
                launch_pattern_matching_kernel(ctx, gpu_task_range);
                hipEventRecord(event);
                log("node %d gpu kernel launched. [%d, %d)\n", node, std::get<0>(gpu_task_range), std::get<1>(gpu_task_range));
            }
        } else { // poll gpu task state
            auto result = hipEventQuery(event);
            if (hipErrorNotReady == result)
                continue;
            
            assert(hipSuccess == result);
            hipDeviceSynchronize();

            unsigned long long sum;
            gpu_check( hipMemcpyFromSymbol(&sum, HIP_SYMBOL(dev_sum), sizeof(sum)) );
            node_ans += sum;
            gpu_ans  += sum;
            gpu_working = false;
            workers_idle[0] = true;
        }
    }

    free_gpu_context(ctx);
}

// other threads are workers
void worker_loop(Graph* g, const Schedule_IEP& sched, int node)
{
    // prepare data structures for pattern matching
    auto ans_buffer = new int[sched.in_exclusion_optimize_vertex_id.size()];
    auto vertex_sets = new VertexSet[sched.get_total_prefix_num()];
    VertexSet partial_embedding, tmp_set;
    partial_embedding.init();

    int thread_id = omp_get_thread_num();
    while (!workers_should_stop) {
        TaskStatus task_range;
        bool idle = !update_task_range(task_range, CPU_WORKER_TASK_GRANULARITY);
        if (idle) {
            workers_idle[thread_id] = true;
            continue;
        }
        workers_idle[thread_id] = false;
        
        int task_begin, task_end;
        std::tie(task_begin, task_end) = task_range;
        log("node %d thread %d do work [%d, %d)\n", node, thread_id, task_begin, task_end);

        uint64_t ans = 0;
        for (int i = task_begin; i < task_end; ++i)
            ans += g->pattern_matching_edge_task(sched, i, vertex_sets, partial_embedding, tmp_set, ans_buffer);
        node_ans += ans;
    }

    // release resources
    delete[] ans_buffer;
    delete[] vertex_sets;
}

void test_cuda_event()
{
    hipEvent_t event;
    hipEventCreate(&event);
    spin_kernel<<<1, 32>>>(1000000000); // ~ 1s, 1e9
    hipEventRecord(event);
    while (true) {
        auto result = hipEventQuery(event);
        if (hipSuccess == result)
            break;
        if (hipErrorNotReady == result) {
            printf("waiting for device...\n");
            usleep(100000);
        } else {
            printf("oops.. %s\n", hipGetErrorString(result));
            break;
        }
    }
}

int main(int argc, char* argv[])
{
    // load graph & build schedule
    if (argc < 4) {
        fprintf(stderr, "Usage: %s graph_file [ignored] pattern_string\n", argv[0]);
        return 0;
    }

    Graph *g;
    DataLoader D;
    if (!D.fast_load(g, argv[1])) {
        fprintf(stderr, "Unable to load graph from %s\n", argv[1]);
        return 0;
    }
    g->build_reverse_edges();

    std::string pattern_str = argv[3];
    Pattern p(int(sqrt(pattern_str.length())), pattern_str.c_str());

    bool pattern_valid;
    Schedule_IEP schedule{p, pattern_valid, 1, 1, true, g->v_cnt, g->e_cnt, g->tri_cnt};
    if (!pattern_valid) {
        fprintf(stderr, "invalid pattern %s\n", argv[3]);
        return 0;
    }

    // MPI initialization
    int comm_sz, node;
    MPI_Init(&argc, &argv);
    MPI_Comm_size(MPI_COMM_WORLD, &comm_sz);
    MPI_Comm_rank(MPI_COMM_WORLD, &node);
    fprintf(stderr, "size = %d rank = %d\n", comm_sz, node);

    // initialize global work states
    nr_tasks = g->e_cnt;
    int nr_threads = omp_get_max_threads();
    workers_idle = new bool[nr_threads] {true};
    // init task_status, global_cur_task
    // warn: make sure nr_tasks >= comm_sz * INITIAL_NODE_TASKS
    int initial_task = node * INITIAL_NODE_TASKS;
    task_status = std::make_tuple(initial_task, initial_task + INITIAL_NODE_TASKS);
    global_cur_task = comm_sz * INITIAL_NODE_TASKS;

    using std::chrono::system_clock;
    auto t1 = system_clock::now();
    #pragma omp parallel
    {
        int thread_id = omp_get_thread_num();
        if (thread_id == 0) {
            scheduler_loop(g, schedule, node, comm_sz);
        } else {
            worker_loop(g, schedule, node);
        }
        log("node %d thread %d finish.\n", node, thread_id);
    }
    auto t2 = system_clock::now();
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    
    if (node == 0) {
        auto final_ans = (global_ans + node_ans) / schedule.get_in_exclusion_optimize_redundancy();
        printf("final answer = %ld\n", final_ans);
        printf("time = %g seconds\n", elapsed.count() / 1e6);
    }
    printf("node %d node_ans = %ld gpu_ans = %ld\n", node, uint64_t(node_ans), gpu_ans);
    MPI_Finalize();
    return 0;
}
